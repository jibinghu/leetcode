#include "hip/hip_runtime.h"
/***************************************************************************
 * 
 *  Copyright (C) 2024.10.8 ISCAS
 *  All rights reserved.
 * 
 *  File Name: Interview.cu
 *  Description: For ISCAS Use.
 * 
 *  This code, or any portion thereof, may not be reproduced, distributed,
 *  or transmitted in any form or by any means, including photocopying,
 *  recording, or other electronic or mechanical methods, without the prior
 *  written permission of the owner.
 * 
 *  Unauthorized use of this code or any part of it may result in legal
 *  action, and the owner reserves the right to pursue legal remedies to
 *  the fullest extent allowed by law.
 * 
 *  Contact Information: 544575367@qq.com
 * 
 ***************************************************************************/

#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define threadsPerBlock 256
#include <iostream>
#include <vector>
using namespace std;
// ========================= reduce =====================
// naive reduce
__global__ void reduce (double *din, double *dout, int n) {
    int tid = threadIdx.x;
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ double sdata [threadsPerBlock];
    if (id < n)
        sdata[tid] = din[id];
    __syncthreads();
    
    for (int s = 1; s < threadsPerBlock; s =  s << 1) {
        //s=1.就是0 2 4线程活着，把135写到自己
        if (tid % (2*s) == 0 && tid + s < threadsPerBlock) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    //相比起来下面的更好，但这是逐渐变好的版本
    //一开始的时候 活着的线程数量更多，但后面活的少了，反而都是在不同warp里的
    //比如倒数第二次操作，0号去拿64，127去拿192的数，一个warp活跃的线程很少

    // for (int stride = threadsPerBlock / 2; stride >= 1; stride /= 2) {
    //     __syncthreads();
    //     if (tid < stride) {
    //         //前半线程把后半对应位置的数加到自己身上，
    //         //当stride小于32，也就是16开始，会bank冲突吗？
    //         //也没有，取时，只有前16线程取数，写时也是各自写
    //         sdata[tid] += sdata[tid+stride];
    //     }
    // }
    // __syncthreads();
    if (tid == 0) {
        dout[blockIdx.x] = sdata[0];
    }
    
}

// no warp divergence
__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;  // tid号线程要负责的数组元素的位置
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();

    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){     // 偶数线程work，
        //没看出好在哪？？？？？？？？？？？？？？？就是后面的虽然没用了但是还是工作
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}
// dim3 gridSize ((N+255)/256)
// dim blockSize 256
// no bank conflict
//从这里开始看上面两个太low了
__global__ void kernel2(double* arr, double* out, int N){
    __shared__ double sdata[threadsPerBlock];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    sdata[tid] = arr[i];
    __syncthreads();
    // 优化1：这里后面32不用同步，可以使用寄存器通信
    // 优化2：可以让一个线程处理多个数据
    //for (int stride = threadsPerBlock / 2; stride >= 1; stride /= 2) {
    //这两句有细微的区别
    for (int s = threadsPerBlock / 2; s > 0; s = s >> 1) {

        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    } 
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}


// 装逼版本1 
__global__ void reduce3(double * d_in, double * d_out, int total_num) {
    int tid = threadIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ double sdata [threadsPerBlock];
    sdata[tid] = d_in[i];
    __syncthreads();
    for (int s = threadsPerBlock / 2; s > 32; s = s >> 1) { // 32 就结束
        if (tid < s && i + s < total_num) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }

    if (tid < 32) { // 因为同一个warp所以不用__syncthreads同步
    //不做条件判断，每一步都全warp一起做 避免
        sdata[tid] += sdata[tid+32];
        sdata[tid] += sdata[tid+16];
        sdata[tid] += sdata[tid+8];
        sdata[tid] += sdata[tid+4];
        sdata[tid] += sdata[tid+2];
        sdata[tid] += sdata[tid+1];
        //每一步的取和写都没有bankconflict
    }
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }

}

////////////////////////------------=============================CURRENt
#define BLOCK_SIZE 256
// 装逼版本2 
template<typename T>
__global__ void reduce4(T * d_in, T * d_out, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    __shared__ T sdata [BLOCK_SIZE];
    sdata[tid] = d_in[i];
    __syncthreads();
    for (int s = BLOCK_SIZE / 2; s > 32; s = s >> 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    T sum = sdata[tid]; //得先取回到线程寄存器
    if (tid < 32) { // 使用shfl操作直接代替shared memory 
        //shfl只能是warp内使用，并且是寄存器通信
        //上一个版本仍是直接操作shared mem，那就不能用shfl
        
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
    }
    if (tid == 0) {
        d_out[blockDim.x] = sdata[0];
    }
}

// =================== reduce 2D ======================
// (M,N) -> (M,1)   首先 (M,N) -> (M,N/256) 再在CPU或者GPU上进行 (M,N/256) -> (M,1)
// gridSize&blockSize <<<(N/256,M), 256>>>
// 分block时，blockIdx.y指第几行，blockIdx.x指每行里，第几个256的块
__global__ void reduce2D_1(double * d_in, double * d_out, int N, int M) {
    int mid = blockIdx.y;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    __shared__ sdata [BLOCK_SIZE];
    sdata[tid] = d_in[mid][i];
    __syncthreads();
    for (int s = BLOCK_SIZE / 2; s >= 32; s = s >> 1) {
        if (tid < s && i + s < N) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    double sum = sdata[tid];
    if (tid < 32) { // 使用shfl操作直接代替shared memory 
        //shfl只能是warp内使用因此上面没有办法用shfl
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
    }
    if (tid == 0) {
        d_out[mid][blockDim.x] = sdata[0];
    }
}

// (M,N) -> (1,N)   首先 (M,N) -> (M/256,N) 再在CPU或者GPU上进行 (M/256,N) -> (1,N)
// gridSize&blockSize <<<(M/256,N), 256>>>
__global__ void reduce2D_2(double * d_in, double * d_out, int N, int M) {
    int nid = blockIdx.y;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    __shared__ sdata [BLOCK_SIZE];
    sdata[tid] = d_in[i][nid];
    //不太好的是 直接就转换了下上一种方法的行列关系
    //同个warp取得数据是在列上连续
    //从global取数都没法连续，如果din是按列存的那还好
    __syncthreads();
    for (int s = BLOCK_SIZE / 2; s >= 32; s = s >> 1) {
        if (tid < s && i + s < M) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    double sum = sdata[tid];
    if (tid < 32) { // 使用shfl操作直接代替shared memory 
        //shfl只能是warp内使用因此上面没有办法用shfl
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
    }
    if (tid == 0) {
        d_out[blockDim.x][nid] = sdata[0];
    }
}

// 当时来自阿里的追问
// Q1: 当 M 本身就小于 256 怎么办
// A1: 减少BLOCKSIZE,直到为32为止(BLOCKSIZE小于WARPSIZE明显不合适)
//想想 M行 所有的行reduce到一行，确实可能行很少
//我觉得上面每行连续的，都分到不同的block确实很不好

// Q2: 当M真的小于 32 怎么办
// A2: 修改代码 (目前只想到了暴力reduce)

// (M,N) -> (1,N) M < 32
// gridSize & blockSize <<<N/256, 256>>>
//为啥一定要256 我觉得32就更好，block起得多 多用sm硬件
//或者就每个warp负责一列，利用shfl可以比较好的把一列的数据进行reduce
__global__ void reduce2D_3(double * d_in, double * d_out, int N, int M) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    for (int j = 0; j < M; j++) {
        d_out[i] += d_in[j][i];
    }
}


// =================== findDiff ======================
// 在一群相同的数中找出唯一不同的数字，基本思路是前后比较，优化就是上shared memory，或者直接寄存器通信

//相同的数字异或会没掉，如果总共的数字数量是奇数，那相同数字有偶数个，一异或就变0，干脆就所有数字异或
//如果是偶数个 就在发现有俩数异或为0时记下来，最后thd0多异或他一次
//这里不用这么麻烦，或许下面这个方法真的还可以，但是操作有点蠢

__global__ void findDiff(int * d_in, int *d_out, int total_num) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int front_id = (id + total_num - 1) % total_num;
    int next_id = (id + total_num + 1) % total_num;
    if (d_in[id] != d_in[front_id] && d_in[id] != d_in[next_id]) {
        d_out = d_in;
    }
}

// =================== DGEMM ======================
// naive GEMM
__global__ void DGEMM(double alpha, double beta, double * d_A, double * d_B, double * d_C, int m, int n, int k, int lda, int ldb, int ldc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    double tmp = 0;
    //写太烂了 两个i是咋回事啊笑死
    for (int i = 0; i < k; i++) {
        tmp += d_A[i*lda+k] * d_B[k*ldb+j];
    }
    d_C[i*ldc+j] = alpha * tmp + beta * d_C[i*ldc+j];
}


// 使用shared memory且没有bank conflict的GEMM
// dim3 blcokSize (32,32)
// dim3 gridSize (M/32, N/32)
#define BLOCK_SIZE 32
__global__ void DGEMM3(double alpha, double beta, double * d_A, double * d_B, double * d_C, int m, int n, int k, int lda, int ldb, int ldc) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x & 31;  //对32取余
    int ty = threadIdx.x >> 5;  //除32.同warp的 取得行也一样
    //block的每个线程都对shared mem上的tile进行1对1
    #define A(i,j) *((d_A) + (i) + (lda)*(j))
    #define B(i,j) *((d_B) + (i) + (ldb)*(j))
    #define C(i,j) *((d_C) + (i) + (ldc)*(j))
    #define sA(i,j) *((s_A) + (i) + (BLOCK_SIZE) *(j))  //j是行索引，i是列索引
    #define sB(i,j) *((s_B) + (i) + (BLOCK_SIZE) *(j))
    __shared__ double s_A[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double s_B[BLOCK_SIZE*BLOCK_SIZE];
    d_A += bx * BLOCK_SIZE; //在
    d_B += by * BLOCK_SIZE * ldb;
    d_C += bx * BLOCK_SIZE + by * BLOCK_SIZE * ldc;
    double tmp = 0;
    for (int inner_k = 0; inner_k < k; inner_k += BLOCK_SIZE) {
        sA(tx, ty) = A(tx, ty); 
        sB(ty, tx) = B(tx, ty); // 注意这里 sB(ty,tx) !!!!!!!!在global行存，到sharedmem column-major
        d_A += BLOCK_SIZE * lda;    //A矩阵在按行取？？？如果是列主序这里才是正确得
        d_B += BLOCK_SIZE;
        __syncthreads();
        for (int kk = 0; kk < BLOCK_SIZE; kk++) {
            tmp += sA(tx, kk) * sB(ty, kk); 
        }
        __syncthreads();//之后要取新得sA/B确保计算完成
    }
    C(tx, ty) = alpha * tmp + beta * C(tx, ty);
}


__global__ void DGEMM3(double alpha, double beta, double * d_A, double * d_B, double * d_C, int m, int n, int k, int lda, int ldb, int ldc) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x & 31;  //对32取余
    int ty = threadIdx.x >> 5;  //除32.同warp的 取得行也一样
    //block的每个线程都对shared mem上的tile进行1对1
    #define A(i,j) *((d_A) + (i) + (lda)*(j))
    #define B(i,j) *((d_B) + (i) + (ldb)*(j))
    #define C(i,j) *((d_C) + (i) + (ldc)*(j))
    #define sA(i,j) *((s_A) + (i) + (BLOCK_SIZE) *(j))  //j是行索引，i是列索引
    #define sB(i,j) *((s_B) + (i) + (BLOCK_SIZE) *(j))
    __shared__ double s_A[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double s_B[BLOCK_SIZE*BLOCK_SIZE];
    d_A += bx * BLOCK_SIZE; //说明A的分块是沿着
    d_B += by * BLOCK_SIZE * ldb;
    d_C += bx * BLOCK_SIZE + by * BLOCK_SIZE * ldc;
    double tmp = 0;
    for (int inner_k = 0; inner_k < k; inner_k += BLOCK_SIZE) {
        sA(tx, ty) = A(tx, ty); 
        sB(ty, tx) = B(tx, ty); // 注意这里 sB(ty,tx) !!!!!!!!在global行存，到sharedmem column-major
        d_A += BLOCK_SIZE * lda;    //A矩阵在按行取？？？如果是列主序这里才是正确得
        d_B += BLOCK_SIZE;
        __syncthreads();
        for (int kk = 0; kk < BLOCK_SIZE; kk++) {
            tmp += sA(tx, kk) * sB(ty, kk); 
        }
        __syncthreads();//之后要取新得sA/B确保计算完成
    }
    C(tx, ty) = alpha * tmp + beta * C(tx, ty);
}

// =================== , ======================
// naive transpose
__global__ void transpose1(double * d_in, double * d_out, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < N && y < N) {
        d_out[x*N+y] = d_in[y*N+x]; 
    }
}
// 那就是合并访存，shared memory
// shared memory 合并访存
#define BLOCK_SIZE 32
__global__ void transpose2 (double * d_in , double * d_out, int M, int N) { ////！！！！
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ s_data[BLOCK_SIZE][BLOCK_SIZE+1];
    if (i < M && j < N ) {
        unsigned int index = i * N + j;
        s_data[threadIdx.y][threadIdx.x] = d_in[index]; //shared[内部j][内部i]=d_in[i][j]
    }
    __syncthreads();
    i = blockDim.y * blockIdx.y + threadIdx.x;  //现在各个block的索引都转置了，i还是用来定行号，但现在一行是M个数据了
    j = blockDim.x * blockIdx.x + threadIdx.y;  //现在用来指结果大矩阵里的列
    // 这个是连续的，32 32 连续，可以合并访存
    //写错了吧if (i < M && j < N ) {
    if (i < N && j < M ) {
        unsigned int index = i * M + j;
        d_out[index] = s_data[threadIdx.x][threadIdx.y];    //现在再按照shared mem里的各行的样子直接按位置搬到global
        //threadIdx在就是行！
    }
}

CURREN
// dim3 blockDim((M+31)/32, (N+31)/32);
// dim3 threadDim(32, 32)
// <<<blockDim, threadDim>>>
//blockIdx.x用来指引行
//从A里读时 是coalesced。然后按行放入sharedmem。取时按列取，再按行写回At，这样global的访问都是coalesced的
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;  //指引列号
  int y = blockIdx.y * TILE_DIM + threadIdx.y;  //感觉y指引行号
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  //blockIdx.y在列方向上 定位块 threadIdx.x是在块内的列位置 transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;  //转置后blockIdx.x * TILE_DIM用来指引行
    //为什么width不变啊啊啊啊
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];   //取得是shared mem里对应转置位置的数
     //唯一疑惑width应该不一样啊除非M==N
}

// ======================  GEMV =================================

// 这里M换成N，然后for循环去处理M。
// 同时如果M还是太大了w
template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

// dim3 dimGrid(M/4);
// dim3 dimBlock(32,4); 
// 在问什么呢，什么变成 64 

// 这个哥们在干嘛
__global__ void Sgemv_v0( 
    float * __restrict__ A,
    float * __restrict__ x,
    float * __restrict__ y, 
    const int M,
    const int N) {
    // Block index
    int bx = blockIdx.x;    //在行方向分block，每个block一次负责blockDim.y行*blockDim.x列个的计算

    //按照grid和block的配置，一个block 4 warps，一个warp一行，就不需要shared mem了，warp内使用shfl

    // Thread index
    int tx = threadIdx.x;   // 不可能
    int ty = threadIdx.y;

    const int warp_size=32;  // warp_size 就是 32 !!!!!!!!!!!! 
    int laneId= tx;
    int current_row = blockDim.y * bx + ty;

    if(current_row < M){
        float ans=0;
        int kIteration = N/warp_size;   //n方向做多次

        for(int i=0; i< kIteration; i++){
            int current_col = i*warp_size + laneId;
            ans += A[current_row*N + current_col] * x[current_col];
        }
        //for(int current_col=laneId; current_col<N; current_col+warp_size)
        //我觉得换成这个也行
        ans += __shfl_down_sync(0xffffffff, ans, 16); // 0-16, 1-17, 2-18, etc.
        ans += __shfl_down_sync(0xffffffff, ans, 8);// 0-8, 1-9, 2-10, etc.
        ans += __shfl_down_sync(0xffffffff, ans, 4);// 0-4, 1-5, 2-6, etc.
        ans += __shfl_down_sync(0xffffffff, ans, 2);// 0-2, 1-3, 4-6, 5-7, etc.
        ans += __shfl_down_sync(0xffffffff, ans, 1);// 0-1, 2-3, 4-5, etc.
        if(laneId==0) y[current_row]=ans;
    }
}






// A: [32, 1024]

#define BLOCK_SIZE 256 /// 先写普通的一会再写shfl的
// =================== reduce 2D ======================
// (M,N) -> (M,1)   首先 (M,N) -> (M,N/256) 再在CPU或者GPU上进行 (M,N/256) -> (M,1)
// gridSize&blockSize <<<(N/256,M), 256>>>
// for 循环去做 (M,N/256) -> (M,1) 在GPU上，他是这个意思
// 
__global__ void reduce2D_1(double * d_in, double * d_out, int N, int M) {
    int mid = blockIdx.y;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    __shared__ sdata [BLOCK_SIZE];
    sdata[tid] = d_in[mid][i];
    __syncthreads();
    for (int s = BLOCK_SIZE / 2; s > 0; s = s >> 1) {
        if (tid < s && i + s < N) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        d_out[mid][blockDim.x] = sdata[0];
    }
    // double sum = sdata[tid];
    // if (tid < 32) { // 使用shfl操作直接代替shared memory 
    //     //shfl只能是warp内使用因此上面没有办法用shfl
    //     sum += __shfl_down_sync(0xffffffff, sum, 16);
    //     sum += __shfl_down_sync(0xffffffff, sum, 8);
    //     sum += __shfl_down_sync(0xffffffff, sum, 4);
    //     sum += __shfl_down_sync(0xffffffff, sum, 2);
    //     sum += __shfl_down_sync(0xffffffff, sum, 1);
    // }
    
}

// naive 版本， 
// 优化：1 用一个线程计算多个数据，
// 优化：2 rs开个数组，现在的reduce效率有点慢
__global__ void getPI(double *rs, int total_num) {
    int tid = threadIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = i * 2 + 1;
    // 把n用i替代
    double s = powf(-1, i);
    double tmp = s * 1.0 / n; //// n 
    // *rs += tmp; /// !!!!!!!!!!!!!!!!!!!!!!!!!
    atomicAdd(rs, tmp);
}


//看不懂问题是啥
__global__ void mergeArrays(const int* inputArray1, const int* inputArray2, int* outputArray, int size)
{
    // 这个是navie想法，问题就是atomicAdd性能太差
    
    // threadIdx 在这里，这个是输出的下标
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIndex < size)
    {
        // index1 index2 初始化为0！！！！！！
        int value1 = inputArray1[index1];
        int value2 = inputArray2[index2];
        
        if (value1 < value2) {
            outputArray[threadIndex] = value1;
            atomicAdd(index1, 1);  // 使用atomicAdd
        } else {
            outputArray[threadIndex] = value2;
            atomicAdd(index2, 1);
        }
            
        
    }
}

// dim (32，,32)
// dim (M/32, N/32, ip2)
__global__ void matrixTranspose(float* input, float* output, int ip2, int n, int m) {
    // 定义共享内存
    __shared__ float sharedInput[BLOCK_SIZE][BLOCK_SIZE];

    // 计算当前线程的索引
    int bIndex = blockIdx.z;    //xi咋还有z？？？
    //还好，看起来是每一层的矩阵进行transpose，各层还是一样的
    int nIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int mIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // 计算输入矩阵和输出矩阵中的索引
    // 这里if去了，编译会报错！！！！！！！！！！！！！！！
    // 连续，
    int inputIndex = bIndex * n * m + nIndex * m + mIndex;  //nIndex是转之前的列号，看起来是按列存（每列m个连续存），mIndex行号
    int outputIndex = bIndex * n * m + mIndex * n + nIndex; //nIndex是转职后的行号，mIndex是列号，并且转置后，每列变成n个，这是符合的
    

    // 将输入矩阵元素加载到共享内存中
    //xi并且threadIdx.y原本指示列号，这里变shared里的行，是因为放入shared时，就是放转置后的样子，那就得按着sharedmem里的样子放进目标global
    if (nIndex < n && mIndex < m) {
        sharedInput[threadIdx.y][threadIdx.x] = input[inputIndex];
    }

    // 同步所有线程，确保共享内存加载完成
    __syncthreads();

    // 检查当前线程是否在矩阵的有效范围内
    if (nIndex < n && mIndex < m) {
        // 执行转置操作
        // no 不连续
        output[outputIndex] = sharedInput[threadIdx.x][threadIdx.y];    //xi我觉得不对，threadIdx.y在shreadmem里表示几列
        //写回output，threadIdx.y也应该是列号，
        //但是看了outputIndex= bIndex * n * m + mIndex * n + nIndex
        //nIndex = blockIdx.y * blockDim.y + threadIdx.y，说明threadIdx.y是global中的行号
    }
}


// 
__global__ void reduce2D_1(double * A, double * y, int lda, int n) {
    int mid = blockIdx.y;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    for (int t = 0; t < n / BLOCKSIZE; t++) {

    }
    __shared__ sdata [BLOCKSIZE];
    sdata[tid] = A[mid*lda+i];
    __syncthreads();
    for (int s = BLOCKSIZE / 2; s > 0; s = s >> 1) {
        if (tid < s && i + s < n) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    // double sum = sdata[tid];
    // if (tid < 32) { // 使用shfl操作直接代替shared memory 
    //     //shfl只能是warp内使用因此上面没有办法用shfl
    //     sum += __shfl_down_sync(0xffffffff, sum, 16);
    //     sum += __shfl_down_sync(0xffffffff, sum, 8);
    //     sum += __shfl_down_sync(0xffffffff, sum, 4);
    //     sum += __shfl_down_sync(0xffffffff, sum, 2);
    //     sum += __shfl_down_sync(0xffffffff, sum, 1);
    // }
    if (tid == 0) {
        y[mid * lda + blockDim.x] = sdata[0];
    }
}

//  这个代码有bug，擦刚刚看出来
// 如果最大的数正好在最后一位，tid = 0 tid = 1 data 都会是那个最大值，先不要告诉她这个事情
// 没事，先这样说吧  这个也算比较聪明方法了
// 看他能不能看出来
// 第二小可能会有点问题
__global__ void findMinTwo(const int* arr, int* min1, int* min2) {
    int data = arr[threadIdx.x];
    int localMin1 = data, localMin2 = data;
    // 和那个没关系，就写我这个就可以，不是shared 的问题
    // 我们就寻找狭义的两个数
    // 这里后16线程是一直没有工作，他的意思可能是让
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        int other = __shfl_down_sync(0xFFFFFFFF, data, offset);
        if (other < data) {
            data = other;
        }
    }
    
//xixi有两个localMin来记录的话，不会让另个在同组里的数消失的吧

    4 5 1 3 5 6 7 8
    // 妈的不是说好了32个数组吗。
    // 
    if (threadIdx.x == 0) {
        *min1 = data;
    }

    data = arr[threadIdx.x];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        int other = __shfl_down_sync(0xFFFFFFFF, data, offset);
        if (other < data && other != *min1) {
            data = other;
        }
    }

    if (threadIdx.x == 0) {
        *min2 = data;
    }

    


}
//当全为row major，黎课说mkn循环最佳（从数据局部性，A和B的数据近期访问的都是行数据
//MKN写法不仅可以优化访存，还可以优化程序的ILP。
//在MNK中，后一次K循环的乘积需要累加到前一次K循环的结果上，程序最内层循环存在迭代依赖。
//MKN的最内层循环不存在这样的迭代依赖，可以释放ILP。
void naive_row_major_sgemm_mkn(const float* A, const float* B, float* C, const int M,
    const int N, const int K) {
    int mi = 0;
    int ni = 0;
    int ki = 0;
    for (mi = 0; mi < M; mi ++) {
        for (ki = 0; ki < K; ki ++) {
            for (ni = 0; ni < N; ni ++) {
                C[mi * N + ni] += A[mi * K + ki] * B[ki * N + ni];
            }
        }
    }

}
// A [m,k]
// B [k,n]
// C [m,n]
void naive_row_major_sgemm(const float* A, const float* B, float* C, const int M,
    const int N, const int K) {
    for (int m = 0; m < M; ++m) {
        for (int n = 0; n < N; ++n) {
            T tmp = 0;
            for (int k = 0; k < K; ++k) {
                tmp += A[m * K + k] * B[k * N + n]; 
            }   
        } 
        C[m*N+n] = tmp;  //  // n在里面对于C矩阵是友好的
    }
}

// dim3 blockSize (32, 32)
// dim3 girdSize ((M+31)/32, (N+31)/32)
// 问问题是不是应该用 could 不用 can
// i can improve this implement  by storing A and B in shared memory
// because  both A and B will be reused 
// but for C, it can't be stored in shared memory
// 使用 reduce 进行优化，将每个A和B乘起来，然后reduce，他们乘积的和
__global__ void naive_row_major_sgemm(const float* A, const float* B, float* C, const int M,
    const int N, const int K) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    __shared__ T sA [32*32];
    __shared__ T sB [32*32];
    // 其他都没有问题  是想重新写吗，这个是对的，我已经按照他的给你改好了
    if (i < M && j < N) {
        T tmp = 0;
        // 和右边是一样的！！
        // 淦，他看错了
        for (int innerk = 0; innerk < K; innerk += 32) { // 问一下idx是正确的吗，我感觉这里用Idx
                                                        // i 也是由idx组成的啊
            sA[idx*32 + idy] = A[i*K + inner_k + idy];  // 这里没有用合并访存，用jforM会更好
            sB[idx*32 + idy] = B[(innerk+idx) * N + j]; // megred memory access
            __syncthreads();
            for (int k = 0; k < 32; k++) {
                tmp += sA[idx*32+k] * B[k *32+ idy];  // 这里可以先将B进行transpose
            }
            __syncthreads();
        }
        C[i*N+j] = tmp;
    }
    
       
}

//xixi尝试优化
__global__ void better_naive_row_major_sgemm(const float* A, const float* B, float* C, const int M,
    const int N, const int K) {
    int i = blockIdx.y * blockDim.y;  //行块的起始位置
    int j = blockIdx.x * blockDim.x;  //列块的起始位置
    int idy = threadIdx.y;  //负责行
    int idx = threadIdx.x;  //负责列

    __shared__ T sA [32*32];
    __shared__ T sB [32*32];
    int tiles = (K+31)/32;
    // 其他都没有问题  是想重新写吗，这个是对的，我已经按照他的给你改好了
    if (i+idy < M && i+idx < N) {
        T tmp = 0;
        //k方向遍历获取宽度为32的块
        for (int k_tile = 0; k_tile < tiles; ++k_tile) { 
            if(k_tile*32+idx<N && k_tile*32+idy<N){
                sA[idy*32 + idx] = A[(i+idy)*K + k_tile*32+idx];  // xi，修改后合并访存了
                sB[idy*32 + idx] = B[(k_tile*32+idy)*N + i+ idx]; // 合并访存，而且也没有bank conflict
            }
            __syncthreads();
            for (int k = 0; k < 32; k++) {
                //一个warp的线程，每次会用相同的sA的数据，和sB同一行的不同的32个数，都没有bankconflict
                tmp += sA[idy*32+k] * sB[k *32+ idx];  
            }
            __syncthreads();
        }
        C[(i+idy)*N+j+idx] = tmp;
    }
    
}

/**
 * @brief 
 *  这个实验性的 用一个block计算 长度为 length数组的前缀和，Input为长度为n 的输入数组，只保证一个block计算前缀和
 *  output则是长度为n的Inpute 前缀和 pre_sum 前缀和 sum[i]= a[0]+a[1]+a[2]+..a[i]
 * @tparam index_t 
 * @tparam value_t 
 * @tparam warp_size  =32 
 * @param Input 
 * @param Output 
 * @param length 
 */
 template<typename index_t,typename value_t,int warp_size=32>
 __global__ void pre_sum_block(value_t * Input,value_t *Output,index_t length)
 {
     const int thid = blockDim.x*blockIdx.x + threadIdx.x; // 总的线程
     const int tx=threadIdx.x;
     const int wrapId = tx / warp_size;
     const int wraps =SDIV(blockDim.x,warp_size); // wraps<=32
     const int laneId = tx & (warp_size-1);// 取二进制最后五位，是 threadIdx对32取模的结果。
 
     if(thid>=length) return;
     // 越界
     value_t val = Input[thid]; // 每个线程的 负责一个数据，本地寄存器上
     __shared__ value_t pre_sum_block [32]; // 每个wrap的最后一个前缀和放在上面
     // const int iters = 
     // 计算 wrap内的前缀和
     #pragma unroll 5
     for(int delta=1;delta<warp_size;delta=delta*2) // 因为warp_size=32，否则应该是 delta< log2f(warp_size)
     {
          value_t temp=__shfl_up_sync(0xFFFFFFFF,val,delta,warp_size);
          if (laneId >=delta)
          //有分支
              val += temp;
         
     }
     // wrap是隐式同步的，限制每个wrap单独计算了前缀和
     if( laneId == warp_size-1)
     {
         // 一个wrap最后一个数
         pre_sum_block[wrapId]=val;
     }
     // 对shared memory的数求前缀和 ,wraps肯定是少于32的
     __syncthreads();// block内同步
 
 //给后面block的每个val加上前面的block的前缀和。又是一种前缀和
 //先用一个warp对pre_sum_block进行一个前缀和

     if(tx<warp_size) // 取第一个wrap对pre_sum_block计算
     {
         value_t warp_share_val = tx<wraps ?  pre_sum_block[tx] :0;
         #pragma unroll 5
         for(int delta=1;delta<warp_size;delta=delta*2) // 因为warp_size=32，否则应该是 delta< log2f(warp_size)
         {
             value_t temp=__shfl_up_sync(0xFFFFFFFF,warp_share_val,delta,warp_size);
             if (laneId >=delta)
                 warp_share_val += temp;
         }
 
         if(tx<wraps) 
             pre_sum_block[tx]= warp_share_val; // 每个wrap最后一个前缀和组成共享数组 的前缀和
 
     }
     __syncthreads();// block内同步，因为不同的warp要读share_memory
     if(wrapId>=1)  // 这里是 >=
     {
         //取wrap左边一个数
         val+=pre_sum_block[wrapId-1];
     }
     Output[thid]=val;
 }
 
 __global__ void scan(int *a,int *b,int equal_value, int N)
 {
    extern    __share__ int share_sum[];
    int tid=thread.x+blockIdx.x*blockDim.x;
    int temp1,temp=0;
    int i=0;
    int t_temp;
    int laneid=thread.x&0x1f,warpid=thread.x/warp_size;
    if((tid<N)&&(a[tid]==equal_value))
    {
      temp=1;
    }
    temp1=temp;//作为标记，用来标记是否写入 
  for(i=1;i<warp_size;i*=2)
  {
      t_temp = __shfl_up_sync(0xFFFFFFFF,temp,i,warp_size);
       if(laneid>=i)
       {
          temp+= t_temp;
       }               
   }
      //这里得出每个线程束的前缀和，且最后一个为最大
      if(laneid==(warp_size-1))
      {
          share_sum[warpid]=temp;
      }
      __sychthread();
      if(!tid)
      //每个block的0号线程把每个warp段的前缀和 加到后一个warp上
      {
           for(i=1;i<(N+blockDim.x-1)/warp_size;i++)
          {
              share_sum[i]=share_sum[i]+share_sum[i-1];
          }
      }
     __sychthread();
      if((laneid!=(warp_size-1))&&(warpid>0))
      { //为什么最后一个线程不要加，他也没有前面warp的前缀和啊
          temp+=share_sum[warpid-1];
      }
      __sychthread();
      if(temp1)
      {
          b[temp-1]=tid;
      }
}


//非CUDA=================================================================================================
//======================================================================================================


// ============================= CPU transpose ======================
struct Tensor {
	vector<int> data; // 张量具体数据
	vector<int> shapes; // 张量形状 {10, 10, 12, 30}  张量的初始perm {0,1,2,3} 
};
// 任意维度transpose函数   perm为转置后的排布 {2,1,0,3}
Tensor transpose(Tensor & d_in, const vector<int> & perm) {
	vector<int> t_data = d_in.data;
	vector<int> t_shapes = d_in.shapes;
	int dims = perm.size();
	Tensor d_out = d_in;
	int total_num = 1; // Tensor内数据元素的个数
    for (auto & i : t_shapes) {
		total_num *= i;
	}
    //统计元素总数

    //xi，先理解下tensor转置，比如原先10组12x30的数据，变成了12组10x30的数据，
    //变成将这10组,同一相对位置的30个数据拼在一起。[9,0,0]数据在新位置[0,9,0]
	
    for (int i = 0; i < total_num; i++) { // 依次遍历各个数据寻找其在d_out中的位置
		int next_i = 0;         // d_out中的位置
		int tmp_i = i;          // d_in中的位置
		vector<int> indexs(dims);   // 相对于shapes的indexs
		for (int t = dims-1; t >= 0; t--) { // 求indexs
			indexs[t] = tmp_i % t_shapes[t];    //对每一维取余
			tmp_i = tmp_i / t_shapes[t];    //除以 更小一维的dim，来看在更高一维里他的相对位置
		}
        //比如上述自己的例子t_shapes[10,12,30]，indexs就是记录获得他在各个维度上的位置

        int stride=1;
        for (int t = dims-1; t>= 0; t--) { // 确定目标的位置
			if (t != dims-1) {
				stride *= t_shapes[perm[t+1]];  //比如在计算新的第2维，那要先乘上新的第3维的stride（也就是dim)
			}
			next_i += indexs[perm[t]]*stride;
            //perm[3]=i,则在新维度的第三维的index 为旧维度的第i维的坐标，即indexs[i]
		}
        //xixi我改成上面！！！！
/*
		for (int t = dims-1; t>= 0; t--) { // 确定目标的位置
			if (t != dims-1) {
				next_i *= t_shapes[perm[t+1]];  //比如在计算新的第2维，那要先乘上新的第3维的stride（也就是dim)
			}
			next_i += indexs[perm[t]];

		}
*/
        //xi我感觉得是，新3维坐标+新2维坐标*新3维dim+新1维坐标*新2维dim*新3维dim
        //xi上面的循环感觉错了
		d_out.data[next_i] = d_in.data[i]; 
	}

    //xi完成struct中的成员shape填写
    for (int t = 0; t < dims; t++) {
        d_out.shapes[t] = t_shapes[perm[t]];
    }
	return d_out;	
}



//
#include 

int findMax(vector<int>& nums) {
    int len = nums.size();
    int left = 0;
    int right = nums.size() - 1;
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (nums[mid] < nums[right]) {
            right = mid;
        }
        else {
            left = mid + 1;
        }
    }
    return (left-1)%len;
}


#include <iostream>
#include <vector>
using namespace std;
/// !!!!!!!!!!!!!!!!!!!!!! 函数写上面！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！
int findMax(vector<int>& nums) {
    int len = nums.size();
    int left = 0;
    int right = nums.size() - 1;

    //xi看傻眼这啥？折半？？？
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (nums[mid] < nums[right]) {
            right = mid; /////!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        }
        else {
            left = mid + 1;   ////!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        }
    }
    return (left-1+len)%len;
}
// idx 为5 他写错了
int main() {
	vector<int> vv = {4,5,6,7,8,9,1,2,3};
	cout << findMax(vv) << endl;
} 



// 

void delete_item (float * data, int len, int * len_ans) {
    //是说在len ans原地删除len个元素？
    if (len == 0) {
        return;
    }
    int fast = 1, slow = 1;
    while (fast < len) {
        if (data[fast] != data[fast - 1]) {
            //看样子是删除重复元素
            //那就双指针，把不重复的写在slow上
            data[slow] = data[fast];
            ++slow;
        }
        ++fast;
    }
    *len_ans = slow; /// *********************************************************
}


// n h w ip3 k_h k_w c_out d_w d_h s_h s_w bias   // 12个参数

// k_w_ k_h_ 是膨胀后的卷积核心
// k_w_ = d_w * (k_w-1) + 1
// k_h_ = d_h * (k_h-1) + 1


class Solution {
public:
    vector<int> inorderTraversal(TreeTreeNode* root) {
        vector<int> result;
        stack<TreeTreeNode*> st;
        if (root != NULL) st.push(root);
        while (!st.empty()) {
            TreeTreeNode* TreeNode = st.top();
            if (TreeNode != NULL) {
                st.pop(); // 将该节点弹出，避免重复操作，下面再将右中左节点添加到栈中
                if (TreeNode->right) st.push(TreeNode->right);  // 添加右节点（空节点不入栈）

                st.push(TreeNode);                          // 添加中节点
                st.push(NULL); // 中节点访问过，但是还没有处理，加入空节点做为标记。

                if (TreeNode->left) st.push(TreeNode->left);    // 添加左节点（空节点不入栈）
            } else { // 只有遇到空节点的时候，才将下一个节点放进结果集
                st.pop();           // 将空节点弹出
                TreeNode = st.top();    // 重新取出栈中元素
                st.pop();
                result.push_back(TreeNode->val); // 加入到结果集
            }
        }
        return result;
    }
};



// dp

#include <iostream>
#inlcude <vector>
using namespace std;
int lengthOfLIS(vector<int>& nums) {
    int maxRs = 0;
    int n = (int)nums.size();
    if (n == 0) {
        return 0;
    }
    vector<int> dp(n, 0);
    for (int i = 0; i < n; ++i) {
        dp[i] = 1;
        for (int j = 0; j < i; ++j) {
            if (nums[j] < nums[i]) {
                dp[i] = max(dp[i], dp[j] + 1);
            }
        }
        if (dp[i] > maxRs) {
            maxRs = dp[i];
        }
    }
    return maxRs;
}

int main()
{
	int n;
	int maxRs = 0;
	cin >> n;
	if (n == 0) return 0;
	vector<int> arr;
    arr.clear();///////
	for (int i = 0; i < n; i++) {
		int num;
		cin >> num;
		arr.emplace_back(num);
	}
	vector<int> dp(n, 0);
	for (int i = 0; i < n; ++i) {
        dp[i] = 1;
        for (int j = 0; j < i; ++j) {
            if (arr[j] < arr[i]) {
                dp[i] = max(dp[i], dp[j] + 1);
            }
        }
        if (dp[i] > maxRs) {
            maxRs = dp[i];
        }
    }
	cout << maxRs;
    return 0;
}



// sum 用 long  long ！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！！
pair<int,int> findOne(vector<int> & arr) {
    int n = arr.size() + 1;
    long long sum = (long long)n * (n+1)/2;
    long long mul = 1;
    for (int i = 1; i <= n; i++) {
        mul *= arr[i];
    }
    for (auto num : arr) {
        sum -= num;
        mul /= num;
    }
    
    // int ip1 = (sum-(int)(sqrt(double(sum*sum-4*mul))))/2;
	// int ip2 = (sum+(int)(sqrt(double(sum*sum-4*mul))))/2;
    int ip1 = (ans1-(int)(sqrt(double(ans1*ans1-4*ans2))))/2;
    int ip2 = (ans1+(int)(sqrt(double(ans1*ans1-4*ans2))))/2;
    return {ip1, ip2};
    
}

int main() {
    
    int n;
    cin >> n;
    for ()
}

class Solution {
public:
    vector<string> restoreIpAddresses(string s) {
        vector<string> ans;
        int n = s.length();
        //遍历IP的点可能的位置（第一个点）
        for(int i = 1; i < 4 && i < n - 2; i++){
            //第二个点的位置
            for(int j = i + 1; j < i + 4 && j < n - 1; j++){
                //第三个点的位置
                for(int k = j + 1; k < j + 4 && k < n; k++){
                    //最后一段剩余数字不能超过3
                    if(n - k >= 4)
                        continue;
                    // 从点的位置分段截取 分别为4个ip地址数据
                    //
                    string ip1 = s.substr(0, i);
                    string ip2 = s.substr(i, j - i);
                    string ip3 = s.substr(j, k - j);
                    string ip4 = s.substr(k);
                    //IP每个数字不大于255
                    if(stoi(ip1) > 255 || stoi(ip2) > 255 || stoi(ip3) > 255 || stoi(ip4) > 255)
                        continue;
                    //    排除前导0的情况
                    if((ip1.length() != 1 && ip1[0] == '0') || (ip2.length() != 1 && ip2[0] == '0') ||  (ip3.length() != 1 && ip3[0] == '0') || (ip4.length() != 1 && ip4[0] == '0'))
                        continue;
                    //组装IP地址
                    string temp = ip1 + "." + ip2 + "." + ip3 + "." + ip4;  /// !!!!! 双引号
                    ans.push_back(temp);
                }
            }
        }
        return ans; /// !!!!
    }
};


 template<typename T>
 class smart
 {
 private:
     T* _ptr;
     int* _count; //reference couting
 
 public:
     //构造函数
     smart(T* ptr = nullptr) :_ptr(ptr)
     {
         if (_ptr)
         {
             _count = new int(1);
         }
         else
         {
             _count = new int(0);
         }
     }
 
     //拷贝构造
     smart(const smart& ptr)
     {
         if (this != &ptr)
         {
             this->_ptr = ptr._ptr;
             this->_count = ptr._count;
 
             (*this->_count)++;
         }
     }
 
     //重载operator=
     smart& operator=(const smart & ptr)
     {
         if (this->_ptr == ptr._ptr)
         {
             return *this;
         }
         if (this->_ptr)
         {
             (*this->_count)--;
             if (*this->_count == 0)
             {
                 delete this->_ptr;
                 delete this->_count;
             }
         }
         this->_ptr = ptr._ptr;
         this->_count = ptr._count;
         (*this->_count)++;
         return *this;
     }
 
     //operator*重载
     T& operator*()
     {
         if (this->_ptr)
         {
             return *(this->_ptr);
         }
     }
 
     //operator->重载
     T* operator->()
     {
         if (this->_ptr)
         {
             return this->_ptr;
         }
     }
 
     //析构函数
     ~smart()
     {
         (*this->_count)--;
         if (*this->_count == 0)
         {
             delete this->_ptr;
             delete this->_count;
         }
     }
     //return reference couting
     int use_count()
     {
         return *this->_count;
     }
 };


#include<iostream>
#include<string.h>
using namespace std;

class String {
private:
	char* m_str;
public:
	// 无参构造
	String(const char* str = "") {
		// +1是为了包含\0
		int len = strlen(str) + 1;
		m_str = new char[len];
		strcpy_s(m_str, len, str);
	}
	// 拷贝构造
	String(const String& s) {
		int len = strlen(s.m_str) + 1;
		m_str = new char[len];
		strcpy_s(m_str, len, s.m_str);
	}
	// 析构
	~String() {
		if (m_str) {
			delete[] m_str;
			m_str = nullptr;
		}
	}
	// 赋值
	String& operator=(const String& s) {
		if (*m_str != *s.m_str) {
			if (m_str != nullptr) {
				delete[] m_str;
				m_str = nullptr;
			}
			int len = strlen(s.m_str) + 1;
			m_str = new char[len];
			strcpy_s(m_str, len, s.m_str);
		}
		return *this;
	}
};


#include<iostream>
#include<thread>
#include<mutex>
#include<condition_variable>
#include<queue>
using namespace std;


int main() {
	mutex mx;
	condition_variable cv;
	queue<int> q;
	const int capicity = 5;

	thread producer([&] {
		for (int i = 0; i < 10; i++) {
			unique_lock<mutex>lock(mx);
			cv.wait(lock, [&] {return q.size() <= capicity; });
			cout << "thread: " << this_thread::get_id() << "produce " << i << endl;
			q.push(i);
			cv.notify_all();
		}
		});

	thread consumer([&] {
		while (true) {
			unique_lock<mutex> lock(mx);
			cv.wait(lock, [&] {return !q.empty(); });
			cout << "thread: " << this_thread::get_id() << "consume " << q.front() << endl;
			q.pop();
			cv.notify_all();
		}
		});
	producer.join();
	consumer.join();
	return 0;
}



#include <iostream>
#include <queue>
// 这个好像他刚刚让你写一下
struct TreeNode {
    int value;
    TreeNode* left;
    TreeNode* right;
    TreeNode(int value):
        value(value), left(nullptr), right(nullptr) {}
};  // !!!!!!!!!!!!!!!  ;;;;;;;;;;;;;;;;;;;;
// 这个是可以的。直接写这个就ok
bool isCBT(TreeNode* head) {
    if (head == nullptr) {
        return true;
    }
    std::queue<TreeNode*> qtree; // 队列结构  
    qtree.push(head);
    TreeNode* tmp = nullptr;
    while (tmp = qtree.front()) {  // 将 tmp的左右结点依次入栈  做的是层级遍历！！！！！！！！ 逐层遍历的
        qtree.push(tmp->left);
        qtree.push(tmp->right);
        qtree.pop();   // 当前 tmp 结点出栈
    } 
    while(!qtree.empty()) {  // 上面是有null就停止了（检测到第一个null），但是剩下的null都还在队里
        if (qtree.front() != nullptr) {  // 按理来说，这边的应该都是null，如果有一个非null，说明这边的不是完全二叉树，因为按照这个遍历，只要出现第一个null，后面不可能有东西了
            return false;
        }
        qtree.pop();
    }
    return true;    // if pass the check, is CBT!
}

int main() {
    TreeNode* head1 = new TreeNode(1);
    head1->left = new TreeNode(2);
    head1->right = new TreeNode(3);
    head1->left->right = new TreeNode(4);
    head1->right->right = new TreeNode(5);

    std::cout << "==============CBT Test1==============\n";
    bool iscbt1 = isCBT(head1);
    std::cout << iscbt1 << std::endl;

    TreeNode* head2 = new TreeNode(1);
    head2->left = new TreeNode(2);
    head2->right = new TreeNode(3);
    head2->left->left = new TreeNode(4);
    head2->left->right = new TreeNode(5);
    head2->right->left = new TreeNode(6);

    std::cout << "==============CBT Test2==============\n";
    bool iscbt2 = isCBT(head2);
    std::cout << iscbt2 << std::endl;
    return 0;
}
// 网上没有原题
// 我目前想的是回溯
// 首先取第一位数字， 首先判断 是否在集合中有这个，如果有则先用（这里是一个回溯点，有可能这个数字是不对的） else 取比这个数更大的一个数字，这个情况下不要考虑回溯，其他数字都按照最小的来就可以
//
/// 从高到低，找到第一个数字在arr中不存在（而且还要比这个数字大），如果说在arr不存在，但是没有比这个数字大也是不可以的
int getNum(vector<int> arr, int m) {
    stack<int> ss;
    while (m > 0) {
        ss.push(m%10);
        m = m / 10;
    }
    while (!ss.empty()) { // 这里是stack，是从最高往最低的不是低到高
        int top = ss.top();
        int num = getMax(arr, top); // 获得大于等于top的最小数
        if (num == top) {
            // 这里应该写一个递归函数
            // 再去取下一个最大
        } else {
            // 这里直接就结束了剩下的都取最小的
        }
    }
}

// 

class Solution {
public:
    // 合并任意两个
    ListTreeNode* mergeTwoLists(ListTreeNode *h1, ListTreeNode *h2) {
        if ((!h1) || (!h2)) return h1 ? h1 : h2;
        ListTreeNode head, *tail = &head, *h1Ptr = h1, *h2Ptr = h2;
        while (h1Ptr && h2Ptr) {
            if (h1Ptr->val < h2Ptr->val) {
                tail->next = h1Ptr;
                h1Ptr = h1Ptr->next;
            } else {
                tail->next = h2Ptr;
                h2Ptr = h2Ptr->next;
            }
            tail = tail->next;
        }
        tail->next = (h1Ptr ? h1Ptr : h2Ptr);
        return head.next;
    }
    // 递归形式的reduce
    // zuobiyoubi 是这样的
    ListTreeNode* mergeReduce(vector <ListTreeNode*> &lists, int l, int r) {
        if (l == r) return lists[l];
        if (l > r) return nullptr;
        int mid = (l + r) >> 1;  /// l!!!!!!!!!!!!!!!!!!!!!!!
        return mergeTwoLists(mergeReduce(lists, l, mid), mergeReduce(lists, mid + 1, r));
    }
    //
    ListTreeNode* mergeKLists(vector<ListTreeNode*>& lists) {
        return mergeReduce(lists, 0, lists.size() - 1);
    }
};