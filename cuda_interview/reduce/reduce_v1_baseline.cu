/*
* 总述：
* 通过不断地将多个元素合并成一个元素，从而减少数据量。场景：计算总和、最小值、最大值等。
* 在 GPU 中，通常利用树状结构进行 reduce 操作； 
* 假设给定一个长度为 N 的数组，需要计算所有元素之和。首先将数组分为 m 小份，
* 第一阶段中开启 m 个 block 计算出 m 个 reduce 值；然后在第二阶段用一个 block 将 m 个值再次 reduce
* Adapted by https://github.com/Liu-xiandong/How_to_optimize_in_GPU
*/

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

#define THREAD_PER_BLOCK 1024 // A100 系列的 THREADS 数量限制；

// 最基础版本，仅用于个人学习；
__global__ void reduce (double *d_in, double *d_out, int n) {
    int tid = threadIdx.x;
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ double sdata [THREAD_PER_BLOCK];

    if (id < n)
        sdata[tid] = d_in[id];
    __syncthreads();
    // s 作为递归扩张系数
    for (int s=1; s<THREAD_PER_BLOCK; s=s<<1) {
        // 这里的 tid+s<THREAD_PER_BLOCK 事实上有些鸡肋：
        // 考虑临界值的情况下， s 再扩大一倍的话必然不满足 s<THREAD_PER_BLOCK 条件；
        if (!(tid%(2*s)) && tid+s<THREAD_PER_BLOCK) {
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    // 注意这里将每个 BLOCK 中归约后的值保存到 d_out 中，也就是 d_out 的大小一定是 N/THREAD_PER_BLOCK
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

// 定义 check 函数
bool check(double *out, double *res, int N){
    for(int i=0; i<N/THREAD_PER_BLOCK; i++){
        if(fabs(out[i]-res[i])>1e-6)
            return false;
    }
    return true;
}

// 定义 main 函数
int main(){
    const int N = 1024;
    // 为 input 分配空间
    double *in = (double *)malloc(N*sizeof(double));
    double *d_in;
    hipMalloc((void **)&d_in,N*sizeof(double)) ;
    // 为 output 数组分配空间
    double *out = (double *)malloc(N/THREAD_PER_BLOCK*sizeof(double));
    double *d_out;
    hipMalloc((void **)&d_out, N/THREAD_PER_BLOCK*sizeof(double));
    // 初始化原数组
    for(int i=0; i<N; i++){
        in[i] = 1.0;
    }
    // 将原数组复制到设备
    hipMemcpy(d_in,in,N*sizeof(double),hipMemcpyHostToDevice);
    // 创建 grid/block，实际运行
    dim3 grid(N/THREAD_PER_BLOCK, 1);
    dim3 block(THREAD_PER_BLOCK, 1);
    reduce<<<grid,block>>>(d_in, d_out, N);
    // 复制 output 到主机
    hipMemcpy(out,d_out,N/THREAD_PER_BLOCK*sizeof(double),hipMemcpyDeviceToHost);
    // 创建验证矩阵并验证
    double *res = (double *)malloc(N/THREAD_PER_BLOCK*sizeof(double));
    for(int i=0;i<N/THREAD_PER_BLOCK;i++){
        double cur = 0.0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur += in[i*THREAD_PER_BLOCK+j];
        }
        res[i] = cur;
    }

    if(check(out,res,N))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<N/THREAD_PER_BLOCK;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    free(in);
    free(out);
    free(res);
    hipFree(d_in);
    hipFree(d_out);
}